#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>
#include <sstream>
#include <algorithm> 
#include <ctime>


using namespace std;

typedef struct
{
	char word[20];
	int count;
	int length;
} word_struct;


// GPU method
/* keyLen - Length of work to be processed by each thread
* dataLen - Total size of data
* data - Pointer to the start of the data
* keyword - Pointer to the start of the keyword
*/
__global__ void searchText(char* data, word_struct* outputArray, int dataLen, int numWords)
{
	int threadIndex = blockIdx.x *blockDim.x + threadIdx.x;
	int instances = 0;
	word_struct thisWord;
	thisWord.length = 0;
	thisWord.count = 0;
	// Initialise the Char array to only spaces
	for (int i = 0; i < 20; i++)
	{
		thisWord.word[i] = ' ';
	}

	/* ----------------------- Find Word by Thread Index ------------------------------*/

		int startOfWord = 0; // Holds the index of the start of a word after the space ->' 't'e's't'
		int lengthOfWord = 0; //Holds the length of the word before the next space
		int spacesCount = 0; // Number of spaces, used to calculate if this threads word has been found
		int wordIndex = 0; // Used to walk the keyword as we walk the data array to check for matches
		bool wordFound = false; // Used to identify if the word was found
		// Find the Keyword I'm looking for
		for (int j = 0; j < dataLen - 1; j++)
		{
			if (data[j] == ' ' && data[j + 1] != ' ') {
				spacesCount++;
				// Start of the word has been found
				if (spacesCount == threadIndex) {
					startOfWord = j + 1;
				}
				// End of the word has been found
				if (spacesCount == threadIndex + 1) {
					lengthOfWord = j - startOfWord;
					wordIndex = 0;
					wordFound = true;
					for (int i = startOfWord; i < startOfWord + lengthOfWord; i++) {
						if (i < startOfWord + 19) {
							thisWord.word[wordIndex] = data[i];
							wordIndex++;
						}
					}
					thisWord.length = lengthOfWord;
					break; // Performance Enhancement
				}
			}
		}

		/* ------------------------------------------------------------------------------------*/

		/* ------------------ Find instances of the word in the data set ----------------------*/

		if (wordFound) {
			int keywordIndex = startOfWord;

			for (int j = 0; j < dataLen - 1; j++)
			{
				if (data[j] == data[keywordIndex])
				{
					keywordIndex++;
					if (keywordIndex == startOfWord + lengthOfWord)
					{ // A full word has been found -
						instances++;
						// Start the keyword from origin again
						keywordIndex = startOfWord;
					}

				}
				else { // The current word doesn't match our keyword
					// Start the keyword from origin again
					keywordIndex = startOfWord;
				}

			}

			/* ------------------------------------------------------------------------------------*/

			/* ------------------ Output data to the console ----------------------*/

			thisWord.count = instances;
			if (spacesCount <= numWords) {
				outputArray[spacesCount - 1] = thisWord;
			}


			/* --------------------------------------------------------------------*/
		}
}

/* ------------------- CUDA Error Handler -----------------------/
Handles Error output for all CUDA operations
/---------------------------------------------------------------*/

void _checkCudaError(char* message, hipError_t err) {
	if (err != hipSuccess) {
		fprintf(stderr, message);
		fprintf(stderr, ": %s\n", hipGetErrorString(err));
		system("pause");
		exit(0);
	}
}

/* -------------------------------------------------------------*/

bool word_sorter(word_struct const& lhs, word_struct const& rhs) {
	// Returns the highest count
	return lhs.count > rhs.count;
}

bool compare_words(word_struct const& lhs, word_struct const& rhs) {
	// Returns -1 if words are not even
	return std::strcmp(lhs.word, rhs.word) < 0;
}

bool remove_words(word_struct const& lhs, word_struct const& rhs) {
	// Returns true if the words are exact matches
	return std::strcmp(lhs.word, rhs.word) == 0;
}

int main(int argc, char* argv[])
{

	/* ------------------- Initialisations ---------------------------/
	Initialise variables for the device and host
	/---------------------------------------------------------------*/

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Dynamic allocation of threads based on GPU hardware
	hipDeviceProp_t deviceProperties;
	hipGetDeviceProperties(&deviceProperties, 0);
	int numThreads = deviceProperties.maxThreadsPerBlock;

	char* text_input = (char*)malloc(512 * sizeof(char));

	/* ------------------- Get User Input ---------------------------/
	Read the name of the files for the string input and stopword dictionary
	/---------------------------------------------------------------*/
	printf("Available Files:\n\n");
	printf("Sample.txt : 478\n");
	printf("Macbook_3k_chars.txt : 3008\n");
	printf("Sheffield_Hallam.txt : 15059\n");
	printf("Cappucino.txt : 35307\n");
	printf("CUDA_50k_chars.txt : 51615\n");
	printf("GoLang_60k_chars.txt : 60438\n");
	printf("Logitech_100k_chars.txt : 83360\n");
	printf("NVIDIA_150k_chars.txt : 143580\n\n");

	printf("Enter the input file name which has to be searched\n");
	scanf("%s", text_input);
	printf("input = %s", text_input);

	/* ----------------------- Read Files ---------------------------/
	Read the input string and the stopwords dictionary from a file.
	/---------------------------------------------------------------*/
	FILE *f = fopen(text_input, "r");
	// Find the end of the file
	fseek(f, 0, SEEK_END);
	// Save the file size
	long fsize = ftell(f);
	fseek(f, 0, SEEK_SET);

	// Devices pointer to memory the size of the file
	char *text = (char *)malloc((fsize + 1) * sizeof(char));
	// Read the file into that memory
	fread(text, fsize, 1, f);

	/* ----------------------- Convert String to Vector ---------------------------/
	To allow us to count the words and appropriately allocate the struct space.
	/-----------------------------------------------------------------------------*/

	std::clock_t cpuStart;
	double cpuPreProcessDuration, cpuPostProcessDuration;

	cpuStart = std::clock();

	string stringText;
	stringText.assign(text, fsize);
	// Make lower case
	transform(stringText.begin(), stringText.end(), stringText.begin(), ::tolower);
	// std::replace_if(stringText.begin(), stringText.end(), ::isdigit, ' ');
	// std::replace_if(stringText.begin(), stringText.end(), ::ispunct, ' ');

	// create a stringstream for our text file
	stringstream ss(stringText);
	// Create two vector iterators
	istream_iterator<string> begin(ss);
	istream_iterator<string> end;
	vector<string> vstrings(begin, end);
	// Copy our data from our string into the vector
	std::copy(vstrings.begin(), vstrings.end(), std::ostream_iterator<std::string>(std::cout, "\n"));

	cpuPreProcessDuration = (std::clock() - cpuStart) / (double)CLOCKS_PER_SEC;

	int numElements = vstrings.size();

	printf("\nNumber of Words to process:  %d", numElements);

	int sizeOfWordStructArray = (numElements * sizeof(word_struct));

	printf("\nBytes required to store device response:  %d", sizeOfWordStructArray);

	/* ----------------------------------------------------------------------------- */

	/* ----------------------- Pre Kernel Tasks -----------------------------------/
	Assign and allocate Memory, Blocks and Threads.
	/-----------------------------------------------------------------------------*/

	// Output array on Device
	word_struct* d_wordArray = (word_struct*)malloc(sizeOfWordStructArray);
	// Output array on Host
	word_struct* h_wordArray = (word_struct*)malloc(sizeOfWordStructArray);

	hipMalloc((void**)&d_wordArray, sizeOfWordStructArray);

	printf("\nFile reading complete...");
	fclose(f);

	int noOfBlocks = strlen(text) / numThreads;
	noOfBlocks++;
	printf("\nBlock size = %d\nFilesize = %d\n", noOfBlocks, fsize);

	char* d_text; // Pointer to the text on the device
	// Allocate memory based on length of string * the memory capacity of a char
	hipMalloc((void**)&d_text, strlen(text) * sizeof(char));
	// Copy text into device variable d_text
	hipMemcpy(d_text, text, strlen(text) * sizeof(char), hipMemcpyHostToDevice);

	/* ---------------------------------------------------------------------------- - */

	/* ----------------------------- Kernel Call -----------------------------/
	Call Kernel and report errors sensibly.
	/------------------------------------------------------------------------*/

	// Error handler, prints pre-defined messages to help debugging
	_checkCudaError(
		"Memory Copy To Device",
		hipGetLastError()
		);

	cpuStart = std::clock();
	hipEventRecord(start);

	/* Calls searchText Kernel, with:
	* d_text : pointer to the text input
	* d_wordArray : array to store the number of word structs
	* datalength : How big our dataset is
	*/
	printf("Sending %d elements to the Kernel", numElements);

	searchText << <noOfBlocks, numThreads >> >(d_text, d_wordArray, strlen(text), numElements);


	hipGetLastError();
	// Error handler, prints pre-defined messages to help debugging
	_checkCudaError(
		"kernel launch",
		hipGetLastError()
		);

	hipDeviceSynchronize();
	// Error handler, prints pre-defined messages to help debugging
	_checkCudaError(
		"Synchronisation",
		hipGetLastError()
		);
	// Copy the contents of the device array to our host array
	hipMemcpy(h_wordArray, d_wordArray, sizeOfWordStructArray, hipMemcpyDeviceToHost);
	// Error handler, prints pre-defined messages to help debugging
	_checkCudaError(
		"Memory Copy From Device",
		hipGetLastError()
		);

	hipEventRecord(stop);
	float milliseconds = (std::clock() - cpuStart) / (double)CLOCKS_PER_SEC;


	/* --------------------------------------------------------------------- */

	/* ----------------------- Post Process Data ----------------/
						Prepare the data for output
	/-----------------------------------------------------------*/

	printf("\nNumber of Results: %d", numElements);
	// Time this process
	cpuStart = std::clock();

	// Create a vector to hold all our unique elements
	std::vector<word_struct> uniqueElements;
	// Assign the contents of h_wordArray
	uniqueElements.assign(h_wordArray, h_wordArray + numElements);
	// Sort the elements using the compare_words function
	std::sort(uniqueElements.begin(), uniqueElements.end(), &compare_words);
	// Create an iterator which uses the remove_words function
	vector<word_struct>::iterator newEnd = unique(uniqueElements.begin(), uniqueElements.end(), &remove_words);
	// Call the iterator on the vector to remove non-unique values
	uniqueElements.erase(newEnd, uniqueElements.end());
	// Sort the vector back into count descending order
	std::sort(uniqueElements.begin(), uniqueElements.end(), &word_sorter);

	/* --------------------------------------------------------------------- */


	/* ----------------------- Print Data ----------------------/
					Output the data meaningfully.
	/-----------------------------------------------------------*/

	hipEventElapsedTime(&milliseconds, start, stop);
	cpuPostProcessDuration = (std::clock() - cpuStart) / (double)CLOCKS_PER_SEC;

	printf("\n|------------------------------------------------------|");
	printf("\n|-------------------- Completed! ----------------------|");
	printf("\n|------------------------------------------------------|");
	printf("\n| File Searched: %s                              ", text_input);
	printf("\n|------------------------------------------------------|");
	printf("\n| Number of Blocks Used: %d                              ", noOfBlocks);
	printf("\n| Number of Threads Used: %d                              ", numThreads);
	printf("\n| Words to process: %d                              ", numElements);
	printf("\n|------------------- Time Taken -----------------------|");
	printf("\n| Time Taken to preprocess data: %fms                              ", cpuPreProcessDuration);
	printf("\n| Time Taken to process data on GPU: %fms                            ", milliseconds);
	printf("\n| Time Taken to postprocess data on CPU: %fms                              ", cpuPostProcessDuration);
	printf("\n| Total Time Taken: %fms                              ", cpuPreProcessDuration + milliseconds + cpuPostProcessDuration);
	printf("\n|------------------------------------------------------|");
	printf("\n|-------------------- Top Ten Results! ----------------|");
	printf("\n|------------------------------------------------------|");

	// Number of printed results
	int printed = 0;

	for (int i = 0; printed < 10; ++i)
	{
		// If the words are longer than two characters they are probably not stop words
		if (uniqueElements[i].length > 2) {
			printf("\n|");
			// Print the whole c-string
			for (int j = 0; j < 19; j++) {
				printf("%c", uniqueElements[i].word[j]);
			}
			printf(" : %d", uniqueElements[i].count);
			++printed;
		}
		else {

		}
	}
	printf("\n|------------------------------------------------------|\n\n\n");

	/* -------------------------------------------------------- */

	hipFree(d_text);
	free(text);

	system("pause");
	return 0;
}